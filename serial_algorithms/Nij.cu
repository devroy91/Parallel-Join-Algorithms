# include "csv.h"
#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

int main(){

  io::CSVReader<3> in1("Trainees.csv");
  in1.read_header(io::ignore_extra_column, "empID", "Name", "managerID");

  string name;
  int empid; 
  int managerid;

  vector<string>table1_name;
  vector<int>table1_empid;
  vector<int>table1_managerid;
 
  while(in1.read_row(empid,name,managerid)){
  	table1_empid.push_back(empid);
  	table1_name.push_back(name);
  	table1_managerid.push_back(managerid);
  }

  io::CSVReader<2> in2("Manager.csv");
  in2.read_header(io::ignore_extra_column, "managerID" , "Name");

  vector<string>table2_name;
  vector<int>table2_managerid;
 
  while(in2.read_row(managerid,name)){
  	table2_managerid.push_back(managerid);
  	table2_name.push_back(name);
  }

  return 0;
}