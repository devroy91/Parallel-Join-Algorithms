#include "hip/hip_runtime.h"
#include"csvUtils.h"
#include<bits/stdc++.h>
#include<thrust/sort.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/execution_policy.h>
#define pb push_back
#include<stdio.h>
#include<thrust/binary_search.h>
#include<thrust/sequence.h>

using namespace std;

void process(int key[], Map &tab, string col){
    key[0] = 0;  //for header
    
    int i = 1;
    for(auto it:tab[col]){
        
	key[i] = it.ival;
	i++;
    }
}

__global__
void countSort(int *key, int *start, int len){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < len){
	int index = key[idx];
        atomicAdd(&start[index], 1);
    }
    
}
void join(int *key1, int *key2, string tab1[], vector<string> &tab2, thrust::host_vector<int>start, int lenTab2, string col){
	
    string lout="";
    lout +=tab1[0]+","+tab2[0]+"\n";
    for(int i = 1; i < lenTab2; i++){
	int val = key2[i];
        int pos = i;
	int s = start[val-1];
	int r = start[val];
	for(int j = s; j < r; j++){
            
	    lout +=tab1[j]+","+tab2[pos]+"\n";
	}
    }
    ofstream outp("ij_Parallel.csv");
    outp << lout;
}
int main(){
    csvstream tab1("Trainee1.csv"), tab2("ManagerO.csv");
    Map trainee, mgr;
    vector<string> strainee, smgr;
    
    //read the tables
    trainee = tab1.read_csv(strainee, true);
    mgr = tab2.read_csv(smgr, true);

    //calculate size of tables
    int lenTab1 = strainee.size();
    int lenTab2 = smgr.size();

    //create index
    int key1[lenTab1];
    int key2[lenTab2];
    string strain[lenTab1];
    
    std::copy(strainee.begin(), strainee.end(), strain);
    
    process(key1, trainee, "managerID");    
    process(key2, mgr, "managerID");
    
    thrust::sort_by_key(key1, key1+lenTab1, (char**)strain);
   
    int maxVal = key1[lenTab1 - 1];
    thrust::host_vector<int> values(maxVal + 1);
    thrust::sequence(values.begin(), values.end());
    thrust::host_vector<int> hist(maxVal + 1);
    thrust::upper_bound(key1, key1 + lenTab1, values.begin(), values.end(), hist.begin());
    
    join(key1, key2, strain, smgr, hist, lenTab2, "managerID");

    return 0;
}
