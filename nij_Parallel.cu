#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "csvUtils.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <stdio.h>
using namespace std;
#define pb push_back 
vector< string > header1, header2;
#define CHUNK_SIZE 1000

//entry structure for cuda
typedef struct cudaEntry{
     int idx;
     int value;
}cudaEntry;


Map read_file(string file1, vector<string> &s, vector<string> &header){
	csvstream tinp(file1);
	
	Map tab1 = tinp.read_csv(s, true);
		
	header = tinp.header;
	return tab1;	

}
thrust::host_vector< cudaEntry > process(Map a, string col){
	
	thrust::host_vector< cudaEntry > intCol;
	for(auto row:a[col]){
		cudaEntry e = {row.idx, row.ival};
		intCol.pb(e);
	}
		
	return intCol;
}
		
__global__
void join(cudaEntry *tab1, cudaEntry *tab2, cudaEntry *out, int *count, int lenTab1, int lenTab2){
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int bd = blockDim.x;
	int idx = bx * bd + tx;
	if(idx < lenTab1){
	    int matches = 0;
	    cudaEntry e1 = *(tab1 + idx);
	    int val1 = e1.value;
	    for(int i = 0; i < lenTab2; i++){
		cudaEntry e2 = *(tab2 + i);
		int val2 = e2.value;
		if(val1 == val2){
		    cudaEntry *temp = out+idx+matches;
		    temp->idx = e1.idx;
		    temp->value = e2.idx;
		    ++matches;
		}
	    }
	}
}

__global__
void countSort(cudaEntry *tab1, cudaEntry *tab2, int *count, int lenTab1, int lenTab2){
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int bd = blockDim.x;
	int idx = bx * bd + tx;
	if(idx < lenTab1){
	    int countVal = 0;
	    cudaEntry e1 = *(tab1 + idx);
	    int val1 = e1.value;
	    for(int i = 0; i < lenTab2; i++){
		cudaEntry e2 = *(tab2 + i);
		int val2 = e2.value;
		if(val1 == val2){
		    countVal++;
		}
	    }
	    count[idx] = countVal;
	}
}
void write_file(string filename, cudaEntry *out, vector<string> &trainee, vector<string> &mgr, int totCount, string col){
	Map d;
	ofstream outp(filename);
	string lout = trainee[0];
	lout.pb(',');
	lout+=mgr[0]+'\n';
	for(int i = 0;i<totCount;i++){
		int idx1 = out[i].idx;
		lout+=trainee[idx1];
		lout.pb(',');	
		
		int idx2 = out[i].value;
		lout +=mgr[idx2]+'\n';
	}
	outp<<lout<<endl;
	outp.close();

}

int main(){
	vector<string> trainee, mgr;
	Map h_tab1, h_tab2;
	h_tab1 = read_file("Trainee1.csv", trainee, header1);
	h_tab2 = read_file("ManagerO.csv", mgr, header2);
	thrust::host_vector< cudaEntry > h_col1, h_col2;
	h_col1 = process(h_tab1, "managerID");
	h_col2 = process(h_tab2, "managerID");
	
	int lenTab1 = h_col1.size();
	int lenTab2 = h_col2.size();
	thrust::device_vector< cudaEntry > d_col1, d_col2;
	d_col1 = h_col1;
	d_col2 = h_col2;
	cudaEntry *pd_col1 = thrust::raw_pointer_cast(d_col1.data());
	cudaEntry *pd_col2 = thrust::raw_pointer_cast(d_col2.data());
	dim3 gridDim = (lenTab1 - 1)/CHUNK_SIZE + 1;
	int *d_count, *h_count;
	hipMalloc((void**)&d_count, lenTab1 * sizeof(int));
	h_count = new int[lenTab1];	

	countSort<<< gridDim, CHUNK_SIZE >>>(pd_col1, pd_col2, d_count, lenTab1,  lenTab2);
	hipMemcpy(h_count, d_count, sizeof(int) * lenTab1, hipMemcpyDeviceToHost);
	int lastThreadCount = h_count[lenTab1 - 1];
	//prefix sum
	thrust::exclusive_scan(h_count, h_count+lenTab1, h_count);
	hipMemcpy(d_count, h_count, sizeof(int) * lenTab1, hipMemcpyHostToDevice);
	int totCount = lastThreadCount + h_count[lenTab1 - 1];
	cudaEntry *d_out, *h_out;
	hipMalloc((void **)&d_out, sizeof(cudaEntry) * totCount);
	h_out = new cudaEntry[totCount]; 
	join<<< gridDim, CHUNK_SIZE >>>(pd_col1, pd_col2, d_out, d_count, lenTab1,  lenTab2);
	hipMemcpy(h_out, d_out, sizeof(cudaEntry) * totCount, hipMemcpyDeviceToHost);

	write_file("nij_Parallel.csv", h_out, trainee, mgr, totCount, "managerID");	
		
	hipFree(d_out);
	hipFree(d_count);
}
